#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdlib>
#include <random>
#include <tuple>
#include <vector>

#include <chrono>

#include <iostream>

#include <rnnt.h>

#include "test.h"

template<typename T>
void vector_to_gpu(T*& gpu_space, std::vector<T>& vec, hipStream_t& stream) {
    hipMalloc(&gpu_space, vec.size() * sizeof(T));
    hipMemcpyAsync(gpu_space, vec.data(), vec.size() * sizeof(T), hipMemcpyHostToDevice, stream);
}

template<typename T>
void vector_to_gpu(T*& gpu_space, const T* cpu_space, int len, hipStream_t& stream) {
    hipMalloc(&gpu_space, len * sizeof(T));
    hipMemcpyAsync(gpu_space, cpu_space, len * sizeof(T), hipMemcpyHostToDevice, stream);
}
float* gen_delay_value(int b, int T, hipStream_t& stream){
    float* buffer= new float[b*T];
    for(int i = 0; i < b; i++){
        for(int j=0;j<T;j ++){
            buffer[i*T +j] = float(j)/T;
        }
    }
    float* dev_buff= nullptr;
    hipMalloc(&dev_buff, sizeof(float)*b*T);
    hipMemcpyAsync(dev_buff, buffer, sizeof(float)*b*T, hipMemcpyHostToDevice, stream);
    return dev_buff;
}

bool run_test(int B, int T, int L, int A, int num_threads) {
    std::mt19937 gen(2);

    auto start = std::chrono::high_resolution_clock::now();
    int len = B * T * (L + 1) * A;
    float * acts = genActs(len);
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    std::cout << "genActs elapsed time: " << elapsed.count() * 1000 << " ms\n";

    std::vector<std::vector<int>> labels;
    std::vector<int> sizes;

    for (int mb = 0; mb < B; ++mb) {
        labels.push_back(genLabels(A, L));
        sizes.push_back(T);
    }

    std::vector<int> flat_labels;
    std::vector<int> label_lengths;
    for (const auto& l : labels) {
        flat_labels.insert(flat_labels.end(), l.begin(), l.end());
        label_lengths.push_back(l.size());
    }

    std::vector<float> costs(B*3);

    rnntOptions options{};
    options.maxT = T;
    options.maxU = L + 1;
    options.blank_label = 0;
    options.loc = RNNT_GPU;
    hipStream_t stream;
    hipStreamCreate(&stream);
    options.stream = stream;
    options.num_threads = num_threads;

    float* acts_gpu;
    vector_to_gpu<float>(acts_gpu, acts, len, stream);
    // hipMalloc(&acts_gpu, len * sizeof(float));
    // hipMemcpyAsync(acts_gpu, acts, len * sizeof(float), hipMemcpyHostToDevice, stream);
    float* grads_gpu;
    hipMalloc(&grads_gpu, len * sizeof(float));
    int* label_gpu;
    vector_to_gpu(label_gpu, flat_labels, stream);
    // hipMalloc(&label_gpu, flat_labels.size() * sizeof(int))
    // hipMemcpyAsync(label_gpu, flat_labels.data(), flat_labels.size() * sizeof(int), hipMemcpyHostToDevice, stream);
    int* label_length_gpu;
    vector_to_gpu(label_length_gpu, label_lengths, stream);
    // hipMalloc(&label_length_gpu, label_lengths.size() * sizeof(int));
    // hipMemcpyAsync(label_length_gpu, label_lengths.data(), label_lengths.size() * sizeof(int), hipMemcpyHostToDevice, stream);
    int* input_length_gpu;
    vector_to_gpu(input_length_gpu, sizes, stream);
    // hipMalloc(&input_length_gpu, sizes.size() * sizeof(int));
    // hipMemcpyAsync(input_length_gpu, sizes.data(), sizes.size() * sizeof(int), hipMemcpyHostToDevice, stream);
    float* delay_values=gen_delay_value(B,T, stream);

    size_t gpu_alloc_bytes;
    throw_on_error(get_delay_workspace_size(T, L+1, B,
                                     true,
                                     &gpu_alloc_bytes),
                    "Error: get_workspace_size in run_test");

    std::vector<float> time;
    // for (int i = 0; i < 10; ++i) {
    for (int i = 0; i < 1; ++i) {
        void* rnnt_gpu_workspace;
        hipMalloc(&rnnt_gpu_workspace, gpu_alloc_bytes);

        start = std::chrono::high_resolution_clock::now();
        throw_on_error(compute_rnnt_delay_loss(acts_gpu, grads_gpu,
                                        label_gpu, label_length_gpu,
                                        input_length_gpu,delay_values,
                                        A, B,
                                        costs.data(),
                                        rnnt_gpu_workspace,1.0,1.0,
                                        options),
                        "Error: compute_rnnt_loss (0) in run_test");
        end = std::chrono::high_resolution_clock::now();
        int U=L+1;
        int grad_size= B*T*U;
        float* grads_cpu= new float[grad_size];
        hipMemcpy(grads_cpu, grads_gpu, sizeof(float) * grad_size, hipMemcpyDeviceToHost);
        float max_=-10, min_=10;
        int badnum=0;
        for(int i =0; i < grad_size; i++){
            if(grads_cpu[i] > max_){
                max_=grads_cpu[i];
            } 
            if(grads_cpu[i] < min_){
                min_=grads_cpu[i];
            } 
            if(grads_cpu[i] >50 || grads_cpu[i]< -50) {
                int u = i % U;
                int bt = (i - u) / U;
                int t = bt % T;
                int mb = (bt - t) / T;
                printf("b=%d, t=%d,u=%d, bad grad =%f\n", mb,t, u, grads_cpu[i]);
                badnum++;
            }
        }
        printf("max=%.4f,min=%.4f, badnum=%d\n", max_, min_, badnum);
        hipFree(rnnt_gpu_workspace);
        elapsed = end - start;
        time.push_back(elapsed.count() * 1000);
        std::cout << "compute_rnnt_delay_loss elapsed time: " << elapsed.count() * 1000 << " ms\n";
    }

    hipFree(grads_gpu);
    hipFree(label_gpu);
    hipFree(label_length_gpu);
    hipFree(input_length_gpu);

    float sum = 0;
    //for (int i = 0; i < 10; ++i) {
    for (int i = 0; i < 1; ++i) {
        sum += time[i];
    }
    sum /= time.size();

    float std = 0;
    for (int i = 0; i < 10; ++i) {
        std += (time[i] - sum) * (time[i] - sum);
    }
    std /= time.size();

    std::cout << "average 10 time cost: " << sum << " ms variance: " << std << std::endl;

    float cost = std::accumulate(costs.begin(), costs.end(), 0.);

    free(acts);
    return true;
}

int main(int argc, char** argv) {
    if (argc < 5) {
        std::cerr << "Arguments: <Batch size> <Time step> <Label length> <Alphabet size>\n";
        return 1;
    }

    int B = atoi(argv[1]);
    int T = atoi(argv[2]);
    int L = atoi(argv[3]);
    int A = atoi(argv[4]);
    std::cout << "Arguments: " \
                << "\nBatch size: " << B \
                << "\nTime step: " << T \
                << "\nLabel length: " << L \
                << "\nAlphabet size: " << A \
                << std::endl;
    
    int num_threads = 1;
    if (argc >= 6) {
        num_threads = atoi(argv[5]);
        std::cout << "Num threads: " << num_threads << std::endl;
    }

    run_test(B, T, L, A, num_threads);
}